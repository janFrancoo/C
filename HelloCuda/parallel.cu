#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 10000000

__global__ void vectorAddition(int n, float *vec1, float *vec2, float *out) {
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n)
        out[tid] = a[tid] + b[tid];
}

int main() {
    float *vec1, *vec2, *out;
    float *d_vec1, *d_vec2, *d_out;

    vec1 = (float *) malloc(sizeof(float) * N);
    vec2 = (float *) malloc(sizeof(float) * N);
    out = (float *) malloc(sizeof(float) * N);

    for (int i=0; i<N; i++) {
        vec1[i] = 1.0f;
        vec2[i] = 1.0f;
    }

    hipMalloc((void**) &d_vec1, sizeof(float) * N);
    hipMalloc((void**) &d_vec2, sizeof(float) * N);
    hipMalloc((void**) &d_out, sizeof(float) * N);

    hipMemcpy(d_vec1, vec1, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, vec2, sizeof(float) * N, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((N + blockSize) / blockSize);

    clock_t begin = clock();
    vectorAddition<<<gridSize, blockSize>>>(N, d_vec1, d_vec2, d_out);   // Multiple thread blocks, 256 parallel threads each
    clock_t end = clock();
    double elapsedTime = (double) (end - begin) / CLOCKS_PER_SEC;

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    hipFree(d_vec1);
    hipFree(d_vec2);
    hipFree(d_out);

    free(vec1);
    free(vec2);
    free(out);

    printf("%f", elapsedTime);

    return 0;
}
