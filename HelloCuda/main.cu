#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define N 10000000

/*
void vectorAddition(int n, float *vec1, float *vec2, float *out) {
    for (int i=0; i<n; i++)
        out[i] = vec1[i] + vec2[i];
}
*/

__global__ void vectorAddition(int n, float *vec1, float *vec2, float *out) {
    for (int i=0; i<n; i++)
        out[i] = vec1[i] + vec2[i];
}

int main() {
    float *vec1, *vec2, *out;
    float *d_vec1, *d_vec2, *d_out;

    vec1 = (float *) malloc(sizeof(float) * N);
    vec2 = (float *) malloc(sizeof(float) * N);
    out = (float *) malloc(sizeof(float) * N);

    for (int i=0; i<N; i++) {
        vec1[i] = 1.0f;
        vec2[i] = 1.0f;
    }

    hipMalloc((void**) &d_vec1, sizeof(float) * N);
    hipMalloc((void**) &d_vec2, sizeof(float) * N);
    hipMalloc((void**) &d_out, sizeof(float) * N);

    hipMemcpy(d_vec1, vec1, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, vec2, sizeof(float) * N, hipMemcpyHostToDevice);

    clock_t begin = clock();
    // vectorAddition(N, vec1, vec2, out);
    vectorAddition<<<1,1>>>(N, d_vec1, d_vec2, d_out);
    clock_t end = clock();
    double elapsedTime = (double) (end - begin) / CLOCKS_PER_SEC;

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    hipFree(d_vec1);
    hipFree(d_vec2);
    hipFree(d_out);

    free(vec1);
    free(vec2);
    free(out);

    printf("%f", elapsedTime);

    return 0;
}
